#include "hip/hip_runtime.h"
#include <stdio.h>
#include "kernel1.h"


//extern  __shared__  float s_data[];

////////////////////////////////////////////////////////////////////////////////
//! Weighted Jacobi Iteration
//! @param g_dataA  input data in global memory
//! @param g_dataB  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void k1( float* g_dataA, float* g_dataB, int floatpitch, int width) 
{
   extern __shared__ float s_data[];
    // TODO, implement this kernel below
    
    size_t x = blockIdx.x * blockDim.x + threadIdx.x + 1;
    size_t y = blockIdx.y * blockDim.y + threadIdx.y + 1;
    size_t s_dim_x = blockDim.x + 2;
    size_t tid_x = threadIdx.x;
    size_t tid_y = threadIdx.y;
    
    // edges of the array
    if (x == 0 || y == 0 || x >= (width - 1)) return;
    
    // if true, left edge of block
    if (tid_x == 0) {      
                         
        s_data[tid_y*s_dim_x + tid_x] = g_dataA[(y - 1) * floatpitch + (x-1)];  // store NW
        s_data[(tid_y+ 1) * s_dim_x + tid_x] = g_dataA[y*floatpitch + (x-1)]; // store W
        s_data[(tid_y+2) * s_dim_x + tid_x] = g_dataA[(y + 1)*floatpitch + (x - 1)];// store SW
    }
    
    // right edge of the block
    if (tid_x == (blockIdx.x - 1) || x <= width - 2) {
    
        s_data[tid_y*s_dim_x + threadIdx.x + 2] = g_dataA[(y - 1) * floatpitch + x + 1];    // NE
        s_data[(tid_y+1) * s_dim_x + threadIdx.x + 2] = g_dataA[y * floatpitch + x + 1];    // E
        s_data[(tid_y+2) * s_dim_x + threadIdx.x + 2] = g_dataA[(y + 1) * floatpitch +x+1]; // SE
    }
    
    // else threads are within the block boundaries
    s_data[(tid_y+2)*s_dim_x+tid_x+1] = g_dataA[(y+1)*floatpitch+x];// N
    s_data[(tid_y+1)*s_dim_x+tid_x+1] = g_dataA[y*floatpitch+x]; // itself
    s_data[tid_y * s_dim_x+tid_x+1] = g_dataA[(y-1)*floatpitch+x];    // S
    __syncthreads();
    
g_dataB[y * floatpitch + x] = (
			0.2f * s_data[(tid_y+1) * s_dim_x + tid_x + 1]  +         //itself
			0.1f * s_data[tid_y*s_dim_x + tid_x + 1]        +         //N
			0.1f * s_data[tid_y*s_dim_x + tid_x + 2]        +         //NE
			0.1f * s_data[(tid_y+1) * s_dim_x + tid_x + 2]  +         //E
			0.1f * s_data[(tid_y+2) * s_dim_x + tid_x + 2]  +         //SE
			0.1f * s_data[(tid_y+2) * s_dim_x + tid_x + 1]  +         //S
			0.1f * s_data[(tid_y+2) * s_dim_x + tid_x]      +         //SW
			0.1f * s_data[(tid_y+1) * s_dim_x + tid_x]      +         //W
			0.1f * s_data[tid_y * s_dim_x + tid_x]                      //NW
		   ) * 0.95f;
    
}


//This version of Kernel uses optimization by copying the data into shared memory and hence results in better performance
//Based upon example kernel at https://developer.nvidia.com/cuda-education
__global__ void calculateCFD_V2( float* g_dataA, float* g_dataB, int floatpitch, int width)
{

	float h = 1.0f/(width-1);

	//Current Global ID
	int i = blockDim.y * blockIdx.y + threadIdx.y; // Y - ID
	int j = blockDim.x * blockIdx.x + threadIdx.x; // X - ID
	
	//Current Local ID (lXX --> refers to local ID i.e. inside a block)
	int block_y = threadIdx.y;
	int block_x = threadIdx.x;
	
	// s_XX --> variables refers to expanded shared memory location in order to accomodate halo elements
	//Current Local ID with 1 offset.
	int s_block_y = block_y + 1;
	int s_block_x = block_x + 1;

	// Variable pointing at top and bottom neighboring location
	int s_block_y_prev = s_block_y - 1;
	int s_block_y_next = s_block_y + 1;

	// Variable pointing at left and right neighboring location
	int s_block_x_prev = s_block_x - 1;
	int s_block_x_next = s_block_x + 1;

	extern __shared__ float s_data[];
	
	unsigned int index = (i)* floatpitch + (j) ;

	if( block_y<1 ) // copy top and bottom halo
	{
		//Copy Top Halo Element
		if(blockIdx.y > 0) // Boundary check
			s_data[block_y*blockDim.x+s_block_x] = g_dataA[index - 1 * floatpitch];

		//Copy Bottom Halo Element
		if(blockIdx.y < (gridDim.y-1)) // Boundary check
			s_data[s_block_y*blockDim.x+blockDim.y+ s_block_x] = g_dataA[index + blockDim.y * floatpitch];
  
	}

	if( block_x<1 ) // copy left and right halo
	{
		if( blockIdx.x > 0) // Boundary check
			s_data[block_y*blockDim.x+s_block_x] = g_dataA[index - 1];
		
		if(blockIdx.x < (gridDim.x-1)) // Boundary check
			s_data[block_y*blockDim.x+s_block_x+blockDim.x] = g_dataA[index + blockDim.x];
	}
	
	// copy current location
	s_data[block_y+s_block_x*blockDim.x] = g_dataA[index]; 

	__syncthreads( );

	if( i > 0 && j > 0 && i < (width-1) && j <(floatpitch-1))
		g_dataB[index] = 0.25f * (s_data[s_block_y_prev*blockDim.x+s_block_x] + s_data[s_block_y_next*blockDim.x+s_block_x] + s_data[s_block_y*blockDim.x + s_block_x_prev] 
			+ s_data[s_block_y*blockDim.x+ s_block_x_next] - 4*h*h);
	
}
